#include "hip/hip_runtime.h"
//ll includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "nvml.h"
#include "cuda_functions.h"

// includes, project
//#include "magma.h"
#include "cuda_multi_gemm_unif.cu"
//#include "cuda_add_vec.h"

//My includes
#include "debug_fns.h"
#include "transformations.h"

//switch the comments to toggle debug mode
//#define D
#define D for(;0;)

double get_time( void )
{
    struct timeval t;
    gettimeofday( &t, NULL );
    return t.tv_sec + t.tv_usec*1e-6;
}

__global__ void particles_in_nid(int *fptsmap, double *rfpts, int *ifpts, double *rpart, int *ipart, double *range, int nrf, int nif, int nfpts, int nr, int ni, int n, int lpart, int nelt, int jx, int jy, int jz,int je0, int jrc, int jpt, int jd, int jr, int nid){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n){
        //double *rpart = rpart1 + id * nr;
        //int *ipart = ipart1 + id * ni;
        int ie;
        double xloc = rpart[id*nr+jx];
        double yloc = rpart[id*nr+jy];
        double zloc = rpart[id*nr+jz];
        for(ie = 0; ie < nelt; ie++){
            //double * range = xerange + ie * 6;
            if(xloc >= range[ie*6+0] && xloc <= range[ie*6+1] && yloc >=range[ie*6+2] && yloc <= range[ie*6+3] && zloc >= range[ie*6+4] && zloc <= range[ie*6+5]){
                ipart[id*ni+je0] = ie;
                ipart[id*ni+jrc] = 0;
                ipart[id*ni+jpt] = nid;
                ipart[id*ni+jd] = 1;
                rpart[id*nr+jr] = -1.0 + 2.0*(xloc-range[ie*6+0])/(range[ie*6+1]-range[ie*6+0]);
                rpart[id*nr+jr+1] = -1.0 + 2.0*(yloc-range[ie*6+2])/(range[ie*6+3]-range[ie*6+2]);
                rpart[id*nr+jr+2] = -1.0 + 2.0*(zloc-range[ie*6+4])/(range[ie*6+5]-range[ie*6+4]);
                break;
            }
        }
        if(ie==nelt){
            //point is outside all elements
            int old = atomicAdd(nfpts, 1);
            if(old==lpart){
                printf("error many moving particles\n");
                return;
            }
            fptsmap[old] = id+1;
            //double * rfp = rfpts + old * nrf;
            //int * ifp = ifpts + old * nif;
            for(int i = 0 ; i < nrf; i++)
                rfpts[old*nrf+i] = rpart[id*nr+i];

            for(int i = 0 ; i < nif; i++)
                ifpts[old*nif+i] = ipart[id*ni+i];
        }
    }

}


__global__ void update_particle_location(double *rpart1, int *ipart, double *xdrange1, int *in_part, int *bc_part, int ndim, int nr, int ni, int n, int jx, int jx1, int jx2, int jx3){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n*ndim){
          int i = id/ndim;
          int j = id%ndim;
          int off = i*nr+j;
          double *rpart = rpart1+off;
          double *xdrange = xdrange1+2*j;

          if (rpart[jx] < xdrange[0]){
                if ( (bc_part[1] == 0 && j == 0) || (bc_part[3] == 0 && j == 1) || (bc_part[5] == 0 && j == 2) ){
                     rpart[jx] = xdrange[1] - fabs(xdrange[0] - rpart[jx]);
                     rpart[jx1] = xdrange[1] + fabs(xdrange[0] - rpart[jx1]);
                     rpart[jx2] = xdrange[1] + fabs(xdrange[0] - rpart[jx2]);
                     rpart[jx3] = xdrange[1] + fabs(xdrange[0] - rpart[jx3]);
               }
               else if ( (bc_part[1] != 0 && j == 0) || (bc_part[3] != 0 && j == 1) || (bc_part[5] != 0 && j == 2) ){
                     int old = atomicExch(in_part[i], -1);
               }
         }
         if (rpart[jx] > xdrange[1]){
               if ( (bc_part[1] == 0 && j == 0) || (bc_part[3] == 0 && j == 1) || (bc_part[5] == 0 && j == 2) ){
                    rpart[jx] = xdrange[0] + fabs(xdrange[0] - rpart[jx]);
                    rpart[jx1] = xdrange[0] - fabs(xdrange[0] - rpart[jx1]);
                    rpart[jx2] = xdrange[0] - fabs(xdrange[0] - rpart[jx2]);
                    rpart[jx3] = xdrange[0] - fabs(xdrange[0] - rpart[jx3]);
              }
              else if ( (bc_part[1] != 0 && j == 0) || (bc_part[3] != 0 && j == 1) || (bc_part[5] != 0 && j == 2) ){
                    int old = atomicExch(in_part[i], -1);
              }

         }
   }
}



// __global__ void update_data_if_outflow(double *rpart1, int *ipart1, double *in_part, int *in_part, int ic, int nr, int ir){
//       int id = blockIdx.x*blockDim.x+threadIdx.x;
//       if(id < n*ndim){
//             double *rpart = rpart1+(id*nr);
//             double *ipart = ipart1+(id*ir);
//             int sum = abs(bc_part[1])+abs(bc_part[2])+abs(bc_part[3])+abs(bc_part[4])+abs(bc_part[5])+abs(bc_part[6]);
//             if (sum > 0){
//                   if (in_part[id] == 0){
//                         int old = atomicAdd(nfpts, 1);
//                         for(int k=0;k < nr; k++){
//
//                         }
//                   }
//             }
//       }
//
// }


extern "C" void particles_in_nid_wrapper_(int *fptsmap, double *rfpts, int *ifpts, double *rpart, int *ipart, double *xerange, int *nrf, int *nif, int *nfpts, int *nr, int *ni, int *n, int *lpart, int *nelt, int *jx, int *jy, int *jz,int *je0, int *jrc, int *jpt, int *jd, int *jr, int *nid){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rfpts, *d_rpart, *d_xerange;
    int *d_fptsmap, *d_ifpts, *d_ipart, *d_nfpts;
    if(inCPU){
        hipMalloc(&d_rfpts, lpart[0]*nrf[0]*sizeof(double));
        hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
        hipMalloc(&d_xerange, nelt[0]*6*sizeof(double));
        hipMalloc(&d_fptsmap, lpart[0]*sizeof(int));
        hipMalloc(&d_ifpts, lpart[0]*nif[0]*sizeof(int));
        hipMalloc(&d_ipart, n[0]*ni[0]*sizeof(int));

        hipMalloc(&d_nfpts, sizeof(int));


        hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_xerange, xerange, nelt[0]*6*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_ipart, ipart, n[0]*ni[0]*sizeof(int), hipMemcpyHostToDevice);


        hipMemcpy(d_nfpts, nfpts, sizeof(int), hipMemcpyHostToDevice);


    }
    else{
        d_rfpts = rfpts;
        d_rpart= rpart;
        d_xerange = xerange;
        d_fptsmap = fptsmap;
        d_ifpts = ifpts;
        d_ipart = ipart;
        hipMalloc(&d_nfpts, sizeof(int));
        hipMemcpy(d_nfpts, nfpts, sizeof(int), hipMemcpyHostToDevice);

    }
    int blockSize = 1024, gridSize;
    gridSize = (int)ceil((float)n[0]/blockSize);
    // printf ("print var %d %d %d\n", n[0], jx[0], jy[0]);
    particles_in_nid<<<gridSize, blockSize>>>(d_fptsmap, d_rfpts, d_ifpts, d_rpart, d_ipart, d_xerange, nrf[0], nif[0], d_nfpts, nr[0], ni[0], n[0], lpart[0], nelt[0], jx[0]-1, jy[0]-1, jz[0]-1, je0[0]-1, jrc[0]-1, jpt[0]-1, jd[0]-1, jr[0]-1, nid[0]);
    if(inCPU){
        hipMemcpy(ipart, d_ipart, n[0]*ni[0]*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(nfpts, d_nfpts, sizeof(int), hipMemcpyDeviceToHost);

        if(nfpts[0]>0){
            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

        }
        //free
	hipFree(d_rpart);
        hipFree(d_ipart);
        hipFree(d_xerange);
        hipFree(d_fptsmap);
        hipFree(d_rfpts);
        hipFree(d_ifpts);
    }
    else{
        hipMemcpy(nfpts, d_nfpts, sizeof(int), hipMemcpyDeviceToHost);
//        if(nfpts[0]>0){
//            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
//            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
//            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

//        }
        // printf ("print var 1st %d\n", nfpts);
    }
    hipFree(d_nfpts);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    // printf ("print var 2nd %d\n", nfpts);
    //printf("particles in nid time is %f\n",time*1e-03);

}

extern "C" void update_particle_location_wrapper_(double *rpart, int *ipart, double *xdrange, int *in_part, int *bc_part, int *ndim, int *nr, int *ni, int *n, int *jx, int *jx1, int *jx2,int *jx3){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rpart, *d_xdrange;
    int *d_ipart, *d_in_part;
    int ic = 0;
    if(inCPU){

        hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
        hipMalloc(&d_xdrange, 6*sizeof(double));
        hipMalloc(&d_in_part, n[0]*sizeof(int));
        hipMalloc(&d_ipart, n[0]*ni[0]*sizeof(int));
        hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_xerange, xerange, nelt[0]*6*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_ipart, ipart, n[0]*ni[0]*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_in_part, in_part, n[0]*sizeof(int), hipMemcpyHostToDevice);


  }
  else{
      d_rpart= rpart;
      d_xdrange = xdrange;
      d_in_part = in_part;
      d_ipart = ipart;
      d_bc_part = bc_part;
      hipMalloc(&d_in_part, n[0]*sizeof(int));
      hipMemcpy(d_in_part, in_part, n[0]*sizeof(int), hipMemcpyHostToDevice);

  }
  int blockSize = 1024, gridSize;
  gridSize = (int)ceil((float)n[0]*ndim[0]/blockSize);
  update_particle_location<<<gridSize, blockSize>>>(d_rpart,d_ipart,d_xdrange,d_inpart,d_bc_part,ndim[0],nr[0],ni[0],n[0],jx[0],jx1[0],jx2[0],jx3[0]);
//----------need to confirm with Dr. Tania and Dr. Ranka if there is a way to parallelize the memory update - right now im implementing it in the host
//  gridSize = (int)ceil((float)n[0]/blockSize);
//  updated_data_if_outflow<<<gridSize, blockSize>>>(d_rpart,s_ipart,d_inpart,&ic);
  if(inCPU){
      hipMemcpy(ipart, d_ipart, n[0]*ni[0]*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(in_part, d_in_part, n[0]*sizeof(int), hipMemcpyDeviceToHost);


      //free
      hipFree(d_rpart);
      hipFree(d_ipart);
      hipFree(d_xdrange);
      hipFree(d_in_part);
  }
  else{
      hipMemcpy(in_part, d_in_part, n[0]*sizeof(int), hipMemcpyDeviceToHost);
//        if(nfpts[0]>0){
//            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
//            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
//            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

//        }
      // printf ("print var 1st %d\n", nfpts);
  }
  hipFree(d_in_part);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  // printf ("print var 2nd %d\n", nfpts);
  //printf("particles in nid time is %f\n",time*1e-03);

}
