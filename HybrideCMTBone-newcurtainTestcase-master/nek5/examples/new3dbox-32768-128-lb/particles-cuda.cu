#include "hip/hip_runtime.h"
//ll includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "nvml.h"
#include "cuda_functions.h"

// includes, project
//#include "magma.h"
#include "cuda_multi_gemm_unif.cu"
//#include "cuda_add_vec.h"

//My includes
#include "debug_fns.h"
#include "transformations.h"

//switch the comments to toggle debug mode
//#define D
#define D for(;0;)

double get_time( void )
{
    struct timeval t;
    gettimeofday( &t, NULL );
    return t.tv_sec + t.tv_usec*1e-6;
}

__global__ void particles_in_nid(int *fptsmap, double *rfpts, int *ifpts, double *rpart, int *ipart, double *range, int nrf, int nif, int nfpts, int nr, int ni, int n, int lpart, int nelt, int jx, int jy, int jz,int je0, int jrc, int jpt, int jd, int jr, int nid){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n){
        //double *rpart = rpart1 + id * nr;
        //int *ipart = ipart1 + id * ni;
        int ie;
        double xloc = rpart[id*nr+jx];
        double yloc = rpart[id*nr+jy];
        double zloc = rpart[id*nr+jz];
        for(ie = 0; ie < nelt; ie++){
            //double * range = xerange + ie * 6;
            if(xloc >= range[ie*6+0] && xloc <= range[ie*6+1] && yloc >=range[ie*6+2] && yloc <= range[ie*6+3] && zloc >= range[ie*6+4] && zloc <= range[ie*6+5]){
                ipart[id*ni+je0] = ie;
                ipart[id*ni+jrc] = 0;
                ipart[id*ni+jpt] = nid;
                ipart[id*ni+jd] = 1;
                rpart[id*nr+jr] = -1.0 + 2.0*(xloc-range[ie*6+0])/(range[ie*6+1]-range[ie*6+0]);
                rpart[id*nr+jr+1] = -1.0 + 2.0*(yloc-range[ie*6+2])/(range[ie*6+3]-range[ie*6+2]);
                rpart[id*nr+jr+2] = -1.0 + 2.0*(zloc-range[ie*6+4])/(range[ie*6+5]-range[ie*6+4]);
                break;
            }
        }
        if(ie==nelt){
            //point is outside all elements
            int old = atomicAdd(nfpts, 1);
            if(old==lpart){
                printf("error many moving particles\n");
                return;
            }
            fptsmap[old] = id+1;
            //double * rfp = rfpts + old * nrf;
            //int * ifp = ifpts + old * nif;
            for(int i = 0 ; i < nrf; i++)
                rfpts[old*nrf+i] = rpart[id*nr+i];

            for(int i = 0 ; i < nif; i++)
                ifpts[old*nif+i] = ipart[id*ni+i];
        }
    }

}


__global__ void update_particle_location(double *rpart1, double *xdrange1, int *in_part, int *bc_part, int n, int ndim, int nr, int jx, int jx1, int jx2, int jx3){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n*ndim){
          int i = id/ndim;
          int j = id%ndim;
          int off = i*nr+j;
          double *rpart = rpart1+off;
          double *xdrange = xdrange1+2*j;

          if (rpart[jx] < xdrange[0]){
                if ( (bc_part[1] == 0 && j == 0) || (bc_part[3] == 0 && j == 1) || (bc_part[5] == 0 && j == 2) ){
                     rpart[jx] = xdrange[1] - fabs(xdrange[0] - rpart[jx]);
                     rpart[jx1] = xdrange[1] + fabs(xdrange[0] - rpart[jx1]);
                     rpart[jx2] = xdrange[1] + fabs(xdrange[0] - rpart[jx2]);
                     rpart[jx3] = xdrange[1] + fabs(xdrange[0] - rpart[jx3]);
               }
               else if ( (bc_part[1] != 0 && j == 0) || (bc_part[3] != 0 && j == 1) || (bc_part[5] != 0 && j == 2) ){
                     int old = atomicExch(in_part[i], -1);
               }
         }
         if (rpart[jx] > xdrange[1]){
               if ( (bc_part[1] == 0 && j == 0) || (bc_part[3] == 0 && j == 1) || (bc_part[5] == 0 && j == 2) ){
                    rpart[jx] = xdrange[0] + fabs(xdrange[0] - rpart[jx]);
                    rpart[jx1] = xdrange[0] - fabs(xdrange[0] - rpart[jx1]);
                    rpart[jx2] = xdrange[0] - fabs(xdrange[0] - rpart[jx2]);
                    rpart[jx3] = xdrange[0] - fabs(xdrange[0] - rpart[jx3]);
              }
              else if ( (bc_part[1] != 0 && j == 0) || (bc_part[3] != 0 && j == 1) || (bc_part[5] != 0 && j == 2) ){
                    int old = atomicExch(in_part[i], -1);
              }

         }
   }
}


__global__ void interp_props_part_location(double *rpart, int *ipart, double *vx,double *vy,double *vz,double *t,double *vtrans,double *rep,double *xgll,double *ygll,double *zgll,double *wxgll,double *wygll,double *wzgll,int n, int nr, int ni, int nx1, int nx1r, int jr, int ju0, int je0, int jtemp, int jrho){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < n){
      double x,y,z;
      double bot = 0.0;
      double diff;
      double repdum;
      double bwgtz[nx1];
      double bwgty[nx1];
      double bwgtx[nx1];
      x = rpart[id*nr+jr];
      y = rpart[id*nr+jr+1];
      z = rpart[id*nr+jr+2];
  // barycentric interpolation initialization
      for (int k = 0; k < nx1r; k++){
            diff = z - zgll[k];
            if (fabs(diff) < 1.0e-16){
                  if (diff > 0){
                        diff = 1.0e-16;
                  }
                  else{
                        diff = -1.0e-16;
                  }

            }
            bwgtz[k] = wzgll[k]/diff;
     }
     for (int i = 0; i < nx1r; i++){
           diff = x - xgll[i];
           if (fabs(diff) < 1.0e-16){
                 if (diff > 0){
                       diff = 1.0e-16;
                 }
                 else{
                       diff = -1.0e-16;
                 }

           }
           bwgtx[i] = wxgll[i]/diff;
    }
    for (int j = 0; j < nx1r; j++){
          diff = y - ygll[j];
          if (fabs(diff) < 1.0e-16){
               if (diff > 0){
                      diff = 1.0e-16;
               }
               else{
                      diff = -1.0e-16;
               }

          }
          bwgty[j] = wygll[j]/diff;
   }

   for(int k=0; k<nx1r;k++){
         for(int j=0; j<nx1r; j++){
             repdum = bwgty[j] * bwgtz[k] ;
             for(int i = 0; i<nx1r; i++){
                 rep[k*nx1r*nx1r+j*nx1r+i] = repdum * bwgtx[i];
                 bot = bot + rep[k*nx1*nx1+j*nx1+i];
             }
         }
     }
  for(int k=0; k<nx1r;k++){
        for(int j=0; j<nx1r; j++){
              for(int i = 0; i<nx1r; i++){
                    rep[k*nx1r*nx1r+j*nx1r+i] = rep[k*nx1r*nx1r+j*nx1r+i]/bot;
              }

        }

  }

  // barycentric interpolation initialization ends  //

  int ie = ipart[id*ni+je0];
  double top1 = 0.0, top2 = 0.0, top3 = 0.0, top4 = 0.0, top5 = 0.0;
  int nxyz = nx1*nx1*nx1;
  double *fieldx = vx+ie*nxyz;
  double *fieldy = vy+ie*nxyz;
  double *fieldz = vz+ie*nxyz;
  double *fieldt = t+ie*nxyz;
  double *fieldvtrans = vtrans+ie*nxyz;
  //full interpolation
  if (nx1r == nx1){
        for (int i=0; i< nxyz; i++){
              top1 = top1 + rep[i]*fieldx[i];
              top2 = top2 + rep[i]*fieldy[i];
              top3 = top3 + rep[i]*fieldz[i];
              top4 = top4 + rep[i]*fieldt[i];
              top5 = top5 + rep[i]*fieldvtrans[i];
        }
        rpart[id*nr+ju0] = top1;
        rpart[id*nr+ju0+1] = top2;
        rpart[id*nr+ju0+2] = top3;
        rpart[id*nr+jtemp] = top4;
        rpart[id*nr+jrho] = top5;
  }

  else{
        // reduced barycentric interpolation
        int kk = 0,jj = 0, ii = 0;
        double ijk3,ijk2,ijk1;
        for (int k=0;k < nx1; k+=2){
              for (int j=0;j<nx1;j+=2){
                    for (int i=0;i<nx1;i+=2){
                          top1 = top1+rep[k*nx1*nx1+j*nx1+i]*fieldx[k*nx1*nx1+j*nx1+i];
                          top2 = top2+rep[k*nx1*nx1+j*nx1+i]*fieldy[k*nx1*nx1+j*nx1+i];
                          top3 = top3+rep[k*nx1*nx1+j*nx1+i]*fieldz[k*nx1*nx1+j*nx1+i];
                          top4 = top4+rep[k*nx1*nx1+j*nx1+i]*fieldx[k*nx1*nx1+j*nx1+i];
                          top5 = top5+rep[k*nx1*nx1+j*nx1+i]*fieldx[k*nx1*nx1+j*nx1+i];
                    }
              }
        }
        rpart[id*nr+ju0] = top1;
        rpart[id*nr+ju0+1] = top2;
        rpart[id*nr+ju0+2] = top3;
        rpart[id*nr+jtemp] = top4;
        rpart[id*nr+jrho] = top5;

  }

  }

}

//-----------------------------------------------------------

__global__ void usr_particles_forces_bdf(double *rpart, int n, int nr,int jvol,int jrhop, int jfusr, int jf0,int jrho,int jfqs,int ju0,int jv0, int ja, int jdp, int jre, int jtaup, int jcd){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n){
//          int i = id/ndim;
//          int j = id%ndim;
          double pmass,pmassf,rdum;
          double uvel[3],vvel[3];
          double c0,T0,mu_p,Re_p,M_p,vel_diff;
          pmass = rpart[id*nr+jvol]*rpart[id*nr+jrhop];
          //compute_re_particles
          uvel[0] = rpart[id*nr+ju0];
          uvel[1] = rpart[id*nr+ju0+1];
          uvel[2] = rpart[id*nr+ju0+2];
          vvel[0] = rpart[id*nr+jv0];
          vvel[1] = rpart[id*nr+jv0+1];
          vvel[2] = rpart[id*nr+jv0+2];
          c0 = 120.0;
          T0 = 291.15;
          c_sound = 1000000; //fix this at the host level so that no concurrency issues arise
          rpart[id*nr+ja] = c_sound;
          mu_p = mu_0;
          vel_diff = sqrt(((uvel[0]-vvel[0])*(uvel[0]-vvel[0]))+((uvel[0]-vvel[0])*(uvel[1]-vvel[1]))+((uvel[2]-vvel[2])*(uvel[2]-vvel[2])));
          Re_p = rpart[id*nr+jrho]*rpart[id*nr+jdp]*vel_diff/mu_p;
          M_p = vel_diff/rpart[id*nr+ja];
          rpart[id*nr+jre] = Re_p;
          //compute_re_particles ends

          for (int k =0 ; k < ndim[0] ; k++){
            if (k == 1){
              rpart[id*nr+jfusr+k] = -1;
            }
            else{
              rpart[id*nr+jfusr+k] = 0;
            }
            rdum = 0;
            rdum = rdum+rpart[id*nr+jfusr+k];
            rpart[id*nr+jf0+k] = rdum/pmass;
          }
        }

//---------------------------------------------------------------------------

__global__ void usr_particles_forces_rk3(double *rpart, int n, int nr,int jvol,int jrhop, int jfusr, int jf0,int jrho,int jfqs,int ju0,int jv0, int ja, int jdp, int jre, int jtaup, int jcd){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n){
//          int i = id/ndim;
//          int j = id%ndim;
          double pmass,pmassf,rdum,cd,S_qs;
          double uvel[3],vvel[3];
          double c0,T0,mu_p,Re_p,M_p,vel_diff;
          pmass = rpart[id*nr+jvol]*rpart[id*nr+jrhop];
          pmassf = rpart[id*nr+jvol]*rpart[id*nr+jrho];
          //compute_re_particles
          uvel[0] = rpart[id*nr+ju0];
          uvel[1] = rpart[id*nr+ju0+1];
          uvel[2] = rpart[id*nr+ju0+2];
          vvel[0] = rpart[id*nr+jv0];
          vvel[1] = rpart[id*nr+jv0+1];
          vvel[2] = rpart[id*nr+jv0+2];
          c0 = 120.0;
          T0 = 291.15;
          c_sound = 1000000; //fix this at the host level so that no concurrency issues arise
          rpart[id*nr+ja] = c_sound;
          mu_p = mu_0;
          vel_diff = sqrt(((uvel[0]-vvel[0])*(uvel[0]-vvel[0]))+((uvel[0]-vvel[0])*(uvel[1]-vvel[1]))+((uvel[2]-vvel[2])*(uvel[2]-vvel[2])));
          Re_p = rpart[id*nr+jrho]*rpart[id*nr+jdp]*vel_diff/mu_p;
          M_p = vel_diff/rpart[id*nr+ja];
          rpart[id*nr+jre] = Re_p;
          //compute_re_particles ends

          for (int k =0 ; k < ndim[0] ; k++){
            if (k == 1){
              rpart[id*nr+jfusr+k] = -1;
            }
            else{
              rpart[id*nr+jfusr+k] = 0;
            }
            //----------usr_particles_f_qs_rk3 starts
            cd = 0;
            S_qs = rpart[id*nr+jvol]*rpart[id*nr+jrhop]/rpart[id*nr+jtaup];
            rpart[id*nr+jcd+k] = cd;
            rpart[id*nr+jfqs+k] = S_qs*(uvel[k] - vvel[k]);
            rdum = 0;
            rdum = rdum+rpart[id*nr+jfusr+k];
            rdum = rdum+rpart[id*nr+jfqs+k];
            rpart[id*nr+jf0+k] = rdum/pmass;
          }
        }

__global__ void update_vel_and_pos_bdf(double *rpart, int n, int ndim, int nr,int jvol,int jrhop, int jfusr, int jf0,int jrho,int jfqs,int ju0,int jv0, int ja, int jdp, int jre, int jtaup, int jcd){
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  double s;
  if(id < n*ndim){
    int i = id/ndim;
    int k = id%ndim;
    // move data to previous position
    rpart[i*nr+ju3+k] = rpart[i*nr+ju2+k];
    rpart[i*nr+ju2+k] = rpart[i*nr+ju1+k];
    rpart[i*nr+ju1+k] = rpart[i*nr+ju0+k];
    rpart[i*nr+jv3+k] = rpart[i*nr+jv2+k];
    rpart[i*nr+jv2+k] = rpart[i*nr+jv1+k];
    rpart[i*nr+jv1+k] = rpart[i*nr+jv0+k];
    rpart[i*nr+jx3+k] = rpart[i*nr+jx2+k];
    rpart[i*nr+jx2+k] = rpart[i*nr+jx1+k];
    rpart[i*nr+jx1+k] = rpart[i*nr+jx0+k];



    // solve for velocity
    s = 1/rpart[i*nr+jtaup];
    rhs = s*((alpha[1]*rpart[i*nr+ju1+k])+(alpha[2]*rpart[i*nr+ju2+k])+(alpha[3]*rpart[i*nr+ju3+k])+rpart[i*nr+jf0+k]+(beta[1]*rpart[i*nr+jv1+k])+(beta[2]*rpart[i*nr+jv2+k])+(beta[3]*rpart[i*nr+jv3+k]);
    rpart[i*nr+jv0+k] = rhs/(beta[0]+s);
    rhx = beta[1]*rpart[i*nr+jx1+k]+beta[2]*rpart[i*nr+jx2+k]+beta[3]*rpart[i*nr+jx3+k]+rpart[i*nr+jv0+k];
    rpart[i*nr+jx0+k] = rhx/beta[0];
  }
}

//----------
__global__ void update_vel_and_pos_rk3(double *rpart, double *kv_stage_p, double *kx_stage_p, int n, int ndim, int nr,int jv0,int jv1,int jv2,int jv3,int ju0,int ju1,int ju2,int ju3, int jx0, int jx1, int jx2, int jx3, int jf0, int fmfac){
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if(id < n*ndim){
      int i = id/ndim;
      int k = id%ndim;
      if (stage == 1){

        // move data to previous position
        rpart[i*nr+ju3+k] = rpart[i*nr+ju2+k];
        rpart[i*nr+ju2+k] = rpart[i*nr+ju1+k];
        rpart[i*nr+ju1+k] = rpart[i*nr+ju0+k];
        rpart[i*nr+jv3+k] = rpart[i*nr+jv2+k];
        rpart[i*nr+jv2+k] = rpart[i*nr+jv1+k];
        rpart[i*nr+jv1+k] = rpart[i*nr+jv0+k];
        rpart[i*nr+jx3+k] = rpart[i*nr+jx2+k];
        rpart[i*nr+jx2+k] = rpart[i*nr+jx1+k];
        rpart[i*nr+jx1+k] = rpart[i*nr+jx0+k];
        kv_stage_p[i*12+k] = rpart[i*nr+jv0+k];
        kx_stage_p[i*12+k] = rpart[i*nr+jx0+k];

      }


      kv_stage_p[i*12+stage*3+k] = rpart[i*nr+jf0+k];
      kx_stage_p[i*12+stage*3+k] = rpart[i*nr+jv0+k];


      if (stage == 3){
        rpart[i*nr+jx0+k] = kx_stage_p[i*12+k]+fmfac*(kx_stage_p[i*12+3+k]+4.0*kx_stage_p[i*12+6+k]+kx_stage_p[i*12+9+k]);
        rpart[i*nr+jv0+k] = kv_stage_p[i*12+k]+fmfac*(kv_stage_p[i*12+3+k]+4.0*kv_stage_p[i*12+6+k]+kv_stage_p[i*12+9+k]);

      }
    }
}


// __global__ void update_data_if_outflow(double *rpart1, int *ipart1, double *in_part, int *in_part, int ic, int nr, int ir){
//       int id = blockIdx.x*blockDim.x+threadIdx.x;
//       if(id < n*ndim){
//             double *rpart = rpart1+(id*nr);
//             double *ipart = ipart1+(id*ir);
//             int sum = abs(bc_part[1])+abs(bc_part[2])+abs(bc_part[3])+abs(bc_part[4])+abs(bc_part[5])+abs(bc_part[6]);
//             if (sum > 0){
//                   if (in_part[id] == 0){
//                         int old = atomicAdd(nfpts, 1);
//                         for(int k=0;k < nr; k++){
//
//                         }
//                   }
//             }
//       }
//
// }


extern "C" void particles_in_nid_wrapper_(int *fptsmap, double *rfpts, int *ifpts, double *rpart, int *ipart, double *xerange, int *nrf, int *nif, int *nfpts, int *nr, int *ni, int *n, int *lpart, int *nelt, int *jx, int *jy, int *jz,int *je0, int *jrc, int *jpt, int *jd, int *jr, int *nid){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rfpts, *d_rpart, *d_xerange;
    int *d_fptsmap, *d_ifpts, *d_ipart, *d_nfpts;
    if(inCPU){
        hipMalloc(&d_rfpts, lpart[0]*nrf[0]*sizeof(double));
        hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
        hipMalloc(&d_xerange, nelt[0]*6*sizeof(double));
        hipMalloc(&d_fptsmap, lpart[0]*sizeof(int));
        hipMalloc(&d_ifpts, lpart[0]*nif[0]*sizeof(int));
        hipMalloc(&d_ipart, n[0]*ni[0]*sizeof(int));

        hipMalloc(&d_nfpts, sizeof(int));


        hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_xerange, xerange, nelt[0]*6*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_ipart, ipart, n[0]*ni[0]*sizeof(int), hipMemcpyHostToDevice);


        hipMemcpy(d_nfpts, nfpts, sizeof(int), hipMemcpyHostToDevice);


    }
    else{
        d_rfpts = rfpts;
        d_rpart= rpart;
        d_xerange = xerange;
        d_fptsmap = fptsmap;
        d_ifpts = ifpts;
        d_ipart = ipart;
        hipMalloc(&d_nfpts, sizeof(int));
        hipMemcpy(d_nfpts, nfpts, sizeof(int), hipMemcpyHostToDevice);

    }
    int blockSize = 1024, gridSize;
    gridSize = (int)ceil((float)n[0]/blockSize);
    // printf ("print var %d %d %d\n", n[0], jx[0], jy[0]);
    particles_in_nid<<<gridSize, blockSize>>>(d_fptsmap, d_rfpts, d_ifpts, d_rpart, d_ipart, d_xerange, nrf[0], nif[0], d_nfpts, nr[0], ni[0], n[0], lpart[0], nelt[0], jx[0]-1, jy[0]-1, jz[0]-1, je0[0]-1, jrc[0]-1, jpt[0]-1, jd[0]-1, jr[0]-1, nid[0]);
    if(inCPU){
        hipMemcpy(ipart, d_ipart, n[0]*ni[0]*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(nfpts, d_nfpts, sizeof(int), hipMemcpyDeviceToHost);

        if(nfpts[0]>0){
            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

        }
        //free
	      hipFree(d_rpart);
        hipFree(d_ipart);
        hipFree(d_xerange);
        hipFree(d_fptsmap);
        hipFree(d_rfpts);
        hipFree(d_ifpts);
    }
    else{
        hipMemcpy(nfpts, d_nfpts, sizeof(int), hipMemcpyDeviceToHost);
//        if(nfpts[0]>0){
//            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
//            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
//            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

//        }
        // printf ("print var 1st %d\n", nfpts);
    }
    hipFree(d_nfpts);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
    // printf ("print var 2nd %d\n", nfpts);
    //printf("particles in nid time is %f\n",time*1e-03);

}

extern "C" void update_particle_location_wrapper_(double *rpart, double *xdrange, int *in_part, int *bc_part, int *n, int *ndim, int *nr, int *jx, int *jx1, int *jx2,int *jx3){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rpart, *d_xdrange;
    int *d_in_part;*d_bc_part;
//    int ic = 0;
    if(inCPU){

        hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
        hipMalloc(&d_xdrange, 6*sizeof(double));
        hipMalloc(&d_in_part, n[0]*sizeof(int));
        hipMalloc(&d_bc_part, 6*sizeof(int));
        hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_xerange, xerange, nelt[0]*6*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_in_part, in_part, n[0]*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_bc_part, bc_part, 6*sizeof(int), hipMemcpyHostToDevice);


  }
  else{
      d_rpart= rpart;
      d_xdrange = xdrange;
      d_in_part = in_part;
      d_bc_part = bc_part;
//      hipMalloc(&d_in_part, n[0]*sizeof(int));
//      hipMemcpy(d_in_part, in_part, n[0]*sizeof(int), hipMemcpyHostToDevice);

  }
  int blockSize = 1024, gridSize;
  gridSize = (int)ceil((float)n[0]*ndim[0]/blockSize);
  update_particle_location<<<gridSize, blockSize>>>(d_rpart,d_xdrange,d_inpart,d_bc_part,n[0],ndim[0],nr[0],jx[0],jx1[0],jx2[0],jx3[0]);
//----------need to confirm with Dr. Tania and Dr. Ranka if there is a way to parallelize the memory update - right now im implementing it in the host
//  gridSize = (int)ceil((float)n[0]/blockSize);
//  updated_data_if_outflow<<<gridSize, blockSize>>>(d_rpart,s_ipart,d_inpart,&ic);
  if(inCPU){
      hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(xerange, d_xerange, nelt[0]*6*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(in_part, d_in_part, n[0]*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(bc_part, d_bc_part, 6*sizeof(int), hipMemcpyDeviceToHost);


      //free
      hipFree(d_rpart);
      hipFree(d_bc_part);
      hipFree(d_xdrange);
      hipFree(d_in_part);
  }
  else{
//      hipMemcpy(in_part, d_in_part, n[0]*sizeof(int), hipMemcpyDeviceToHost);
//        if(nfpts[0]>0){
//            hipMemcpy(fptsmap, d_fptsmap, nfpts[0]*sizeof(int), hipMemcpyDeviceToHost);
//            hipMemcpy(rfpts, d_rfpts, nfpts[0]*nrf[0]*sizeof(double), hipMemcpyDeviceToHost);
//            hipMemcpy(ifpts, d_ifpts, nfpts[0]*nif[0]*sizeof(int), hipMemcpyDeviceToHost);

//        }
      // printf ("print var 1st %d\n", nfpts);
  }
//  hipFree(d_in_part);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);
  // printf ("print var 2nd %d\n", nfpts);
  //printf("particles in nid time is %f\n",time*1e-03);

}

extern "C" void interp_props_part_location_wrapper_(double *rpart,int *ipart,double *vx, double *vy, double *vz,double *t, double *vtrans, double *rep, double *xgll, double *ygll, double *zgll, double *wxgll, double *wygll, double *wzgll, int* n, int* nr, int* ni, int* nx1, int* nx1r, int* jr,int* ju0,int* je0,int* jtemp,int* jrho,int* nelt){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rpart,*d_vx,*d_vy, *d_vz, *d_t, *d_vtrans;
    int *d_ipart;
    if(inCPU){

        hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
        hipMalloc(&d_ipart, n[0]*ni[0]*sizeof(int));
        hipMalloc(&d_vx, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double));
        hipMalloc(&d_vy, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double));
        hipMalloc(&d_vz, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double));
        hipMalloc(&d_t, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double));
        hipMalloc(&d_vtrans, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double));
       hipMalloc(&d_rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double));
       hipMalloc(&d_xgll, nx1[0]*sizeof(double));
       hipMalloc(&d_ygll, nx1[0]*sizeof(double));
       hipMalloc(&d_zgll, nx1[0]*sizeof(double));
       hipMalloc(&d_wxgll, nx1[0]*sizeof(double));
       hipMalloc(&d_wygll, nx1[0]*sizeof(double));
       hipMalloc(&d_wzgll, nx1[0]*sizeof(double));
       hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_ipart, ipart, n[0]*ni[0]*sizeof(int), hipMemcpyHostToDevice);
       hipMemcpy(d_vx, vx, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_vy, vy, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_vz, vz, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_t, t, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_vtrans, vtrans, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_rep, rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_xgll, xgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_ygll, ygll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_zgll, zgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_wxgll, wxgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_wygll, wygll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
       hipMemcpy(d_wzgll, wzgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);


  }
  else{
      d_rpart= rpart;
      d_ipart = ipart;
      d_vx = vx;
      d_vy = vy;
      d_vz = vz;
      d_t =t;
      d_vtrans = vtrans;
      hipMalloc(&d_rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double));
      hipMalloc(&d_xgll, nx1[0]*sizeof(double));
      hipMalloc(&d_ygll, nx1[0]*sizeof(double));
      hipMalloc(&d_zgll, nx1[0]*sizeof(double));
      hipMalloc(&d_wxgll, nx1[0]*sizeof(double));
      hipMalloc(&d_wygll, nx1[0]*sizeof(double));
      hipMalloc(&d_wzgll, nx1[0]*sizeof(double));
      hipMemcpy(d_rep, rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_xgll, xgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_ygll, ygll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_zgll, zgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_wxgll, wxgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_wygll, wygll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(d_wzgll, wzgll, nx1[0]*sizeof(double), hipMemcpyHostToDevice);
  }
  int blockSize = 1024, gridSize;
  gridSize = (int)ceil((float)n[0]/blockSize);
  interp_props_part_location<<<gridSize, blockSize>>>(d_rpart,d_vx,d_vy,d_vz,d_t,d_vtrans,d_rep,d_xgll,d_ygll,d_zgll,d_wxgll,d_wygll,d_wzgll,n[0],nr[0],ni[0],nx1[0],nx1r[0],jr[0],ju0[0],je0[0],jtemp[0],jrho[0]);
  if(inCPU){
      hipMemcpy(ipart, d_ipart, n[0]*ni[0]*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vx, d_vx, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vz, d_vz, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(t, d_t, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(vtrans, d_vtrans, nx1[0]*nx1[0]*nx1[0]*nelt[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(rep, d_rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(xgll, d_xgll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(ygll, d_ygll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(zgll, d_zgll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(wxgll, d_wxgll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(wygll, d_wygll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(wzgll, d_wzgll, nx1[0]*sizeof(double), hipMemcpyDeviceToHost);

      //free
      hipFree(d_rpart);
      hipFree(d_ipart);
      hipFree(d_vx);
      hipFree(d_vy);
      hipFree(d_vz);
      hipFree(d_t);
      hipFree(d_vtrans);
     hipFree(d_rep);
     hipFree(d_xgll);
     hipFree(d_ygll);
     hipFree(d_zgll);
     hipFree(d_wxgll);
     hipFree(d_wygll);
     hipFree(d_wzgll);
  }

  else{
       hipMemcpy(rep,d_rep, nx1[0]*nx1[0]*nx1[0]*sizeof(double), hipMemcpyDeviceToHost);

        //free
       hipFree(d_rep);
       hipFree(d_xgll);
       hipFree(d_ygll);
       hipFree(d_zgll);
       hipFree(d_wxgll);
       hipFree(d_wygll);
       hipFree(d_wzgll);

 }
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time, startEvent, stopEvent);

  }

 //---------------------------------------------------------------------------
 extern "C" void usr_particles_forces_bdf_wrapper_(double *rpart, int *n, int* nr,int* jvol,int* jrhop, int* jfusr, int* jf0,int* jrho,int* jfqs,int* ju0,int* jv0, int* ja, int* jdp, int* jre, int* jtaup, int* jcd){

     float time;
     hipEvent_t startEvent, stopEvent;
     hipEventCreate(&startEvent);
     hipEventCreate(&stopEvent);
     hipEventRecord(startEvent, 0);

     bool inCPU = false;
     double *d_rpart;
     if(inCPU){
       hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
       hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
     }
     else{
       d_rpart= rpart;
     }
     int blockSize = 1024, gridSize;
     gridSize = (int)ceil((float)n[0]/blockSize);
     usr_particles_forces_bdf<<<gridSize, blockSize>>>(d_rpart,n[0],nr[0],jvol[0],jrhop[0],jfusr[0],jf0[0],jrho[0],jfqs[0],ju0[0],jv0[0],ja[0],jdp[0],jre[0],jtaup[0],jcd[0]);
     if(inCPU){
         hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
         //free
         hipFree(d_rpart);
     }

     hipEventRecord(stopEvent, 0);
     hipEventSynchronize(stopEvent);
     hipEventElapsedTime(&time, startEvent, stopEvent);
}


//---------------------------------------------------------

//---------------------------------------------------------------------------
extern "C" void usr_particles_forces_rk3_wrapper_(double *rpart, int* n, int* nr, int* jvol,int* jrhop, int* jfusr, int* jf0,int* jrho,int* jfqs,int* ju0,int* jv0, int* ja, int* jdp, int* jre, int* jtaup, int* jcd){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rpart;
    if(inCPU){
      hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
      hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
    }
    else{
      d_rpart= rpart;
    }
    int blockSize = 1024, gridSize;
    gridSize = (int)ceil((float)n[0]/blockSize);
    usr_particles_forces_rk3<<<gridSize, blockSize>>>(d_rpart,n[0],nr[0],jvol[0],jrhop[0],jfusr[0],jf0[0],jrho[0],jfqs[0],ju0[0],jv0[0],ja[0],jdp[0],jre[0],jtaup[0],jcd[0]);
    if(inCPU){
        hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
        //free
        hipFree(d_rpart);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
}


//---------------------------------------------------------
extern "C" void update_vel_and_pos_bdf_wrapper_(double *rpart, int *n, int* nr,int* jvol,int* jrhop, int* jfusr, int* jf0,int* jrho,int* jfqs,int* ju0,int* jv0, int* ja, int* jdp, int* jre, int* jtaup, int* jcd){

    float time;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    bool inCPU = false;
    double *d_rpart;
    if(inCPU){
      hipMalloc(&d_rpart, n[0]*nr[0]*sizeof(double));
      hipMemcpy(d_rpart, rpart, n[0]*nr[0]*sizeof(double), hipMemcpyHostToDevice);
    }
    else{
      d_rpart= rpart;
    }
    int blockSize = 1024, gridSize;
    gridSize = (int)ceil((float)n[0]/blockSize);
    update_vel_and_pos_bdf<<<gridSize, blockSize>>>
    if(inCPU){
        hipMemcpy(rpart, d_rpart, n[0]*nr[0]*sizeof(double), hipMemcpyDeviceToHost);
        //free
        hipFree(d_rpart);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&time, startEvent, stopEvent);
}
